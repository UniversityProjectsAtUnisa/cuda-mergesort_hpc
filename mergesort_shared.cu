
#include <hip/hip_runtime.h>
#include <assert.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#define DATA int
#define MIN(a, b) (a < b ? a : b)
#define SIZE 33554432
#define BLOCKSIZE 8
#define GRIDSIZE SIZE / 2 / BLOCKSIZE
#define SHARED 4

#define CUDA_CHECK(X)                                                     \
    {                                                                     \
        hipError_t _m_cudaStat = X;                                      \
        if (hipSuccess != _m_cudaStat)                                   \
        {                                                                 \
            fprintf(stderr, "\nCUDA_ERROR: %s in file %s line %d\n",      \
                    hipGetErrorString(_m_cudaStat), __FILE__, __LINE__); \
            exit(1);                                                      \
        }                                                                 \
    }

int _is_sorted(DATA *arr, size_t size);
void MergeSortOnDevice(DATA *arr, size_t size);
__global__ void gpu_mergesort(DATA *A, DATA *B, size_t size, size_t width);
__device__ void gpu_bottomUpMerge(DATA *arr1, size_t size1, DATA *arr2, size_t size2, DATA *tmp);

int main(int argc, char **argv)
{
    DATA *arr;
    size_t size = SIZE;
    assert(GRIDSIZE * BLOCKSIZE == SIZE / 2);
    assert(size == 0 || !(size & (size - 1)));

    arr = (DATA *)malloc(size * sizeof(DATA));
    if (arr == NULL)
    {
        fprintf(stderr, "Memory could not be allocated");
        exit(EXIT_FAILURE);
    }

    srand(0);
    for (size_t i = 0; i < size; i++)
    {
        arr[i] = rand(); // TODO: generate with sign and maybe in a range
    }

    MergeSortOnDevice(arr, size);
    assert(_is_sorted(arr, size) == 1);
}

int _is_sorted(DATA *arr, size_t size)
{
    for (size_t i = 0; i < size - 1; i++)
        if (arr[i] > arr[i + 1])
            return 0;
    return 1;
}

void MergeSortOnDevice(DATA *arr, size_t size)
{
    if (size == 0)
        return;

    DATA *dArr, *tmp;

    size_t byteSize = size * sizeof(DATA);
    CUDA_CHECK(hipMalloc(&dArr, byteSize));
    CUDA_CHECK(hipMemcpy(dArr, arr, byteSize, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMalloc(&tmp, byteSize));

    hipEvent_t start, stop;
    CUDA_CHECK(hipEventCreate(&start));
    CUDA_CHECK(hipEventCreate(&stop));
    CUDA_CHECK(hipEventRecord(start, 0));

    //
    // Slice up the list and give pieces of it to each thread, letting the pieces
    // grow bigger and bigger until the whole list is sorted
    //

    DATA *A = dArr,
         *B = tmp;

    int nBlocks = GRIDSIZE;
    int blockSize = BLOCKSIZE;
    for (size_t width = 2; width <= size; width <<= 1)
    {
        // int slices = size / (nBlocks * blockSize * width);

        // Actually call the kernel
        gpu_mergesort<<<nBlocks, blockSize>>>(A, B, size, width);
        hipDeviceSynchronize();
        // gpu_mergesort<<<nBlocks, nThreads / blocksPerGrid>>>(
        //     A, B, size, width, slices, D_threads, D_blocks);

        // Switch the input / output arrays instead of copying them around
        A = A == dArr ? tmp : dArr;
        B = B == dArr ? tmp : dArr;

        if (blockSize > 1)
        {
            blockSize /= 2;
        }
        else
        {
            nBlocks /= 2;
        }
    }

    // MergeSortKernel<<<numBlocks, numThreads>>>(
    //     dArr, size, ceil(size / float(BLOCKSIZE * GRIDSIZE)));

    CUDA_CHECK(hipEventRecord(stop, 0));
    CUDA_CHECK(hipEventSynchronize(stop));
    float elapsed;
    CUDA_CHECK(hipEventElapsedTime(&elapsed, start, stop));
    elapsed = elapsed / 1000.f; // convert to seconds
    CUDA_CHECK(hipEventDestroy(start));
    CUDA_CHECK(hipEventDestroy(stop));

    printf("Kernel elapsed time %fs \n", elapsed);

    CUDA_CHECK(hipMemcpy(arr, A, byteSize, hipMemcpyDeviceToHost));

    CUDA_CHECK(hipFree(dArr));
}

__global__ void gpu_mergesort(DATA *A, DATA *B, size_t size, size_t width)
{
    __shared__ DATA localA[BLOCKSIZE * 2];
    __shared__ DATA localB[BLOCKSIZE * 2];
    unsigned int currentThread = threadIdx.x;
    unsigned int globalIdx = blockIdx.x * blockDim.x + currentThread;

    for (size_t i = 0; i < width; i++)
    {
        localA[currentThread + i] = A[globalIdx + i];
    }

    size_t start = width * currentThread;
    if (start >= size)
        return;

    size_t halfSize = width / 2;

    gpu_bottomUpMerge(localA + start, halfSize, localA + start + halfSize, halfSize, localB + start);
    for (size_t i = 0; i < width; i++)
    {
        A[globalIdx + i] = localA[currentThread + i];
        B[globalIdx + i] = localB[currentThread + i];
    }
}

__device__ void gpu_bottomUpMerge(DATA *arr1, size_t size1, DATA *arr2, size_t size2, DATA *tmp)
{
    size_t i = 0, j = 0;

    while (i < size1 && j < size2)
    {
        if (arr1[i] < arr2[j])
        {
            tmp[i + j] = arr1[i];
            i++;
        }
        else
        {
            tmp[i + j] = arr2[j];
            j++;
        }
    }
    while (i < size1)
    {
        tmp[i + j] = arr1[i];
        i++;
    }
    while (j < size2)
    {
        tmp[i + j] = arr2[j];
        j++;
    }
}