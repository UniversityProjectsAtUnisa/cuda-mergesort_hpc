
#include <hip/hip_runtime.h>
#include <assert.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#define DATA int
#define MIN(a, b) (a < b ? a : b)
#define SIZE 4194304
#define BLOCKSIZE 32
#define TASKSIZE 2
#define GRIDSIZE (SIZE / TASKSIZE / BLOCKSIZE)

#define CUDA_CHECK(X)                                               \
  {                                                                 \
    hipError_t _m_cudaStat = X;                                    \
    if (hipSuccess != _m_cudaStat) {                               \
      fprintf(stderr, "\nCUDA_ERROR: %s in file %s line %d\n",      \
              hipGetErrorString(_m_cudaStat), __FILE__, __LINE__); \
      exit(1);                                                      \
    }                                                               \
  }

int _is_sorted(DATA *arr, size_t size);
void MergeSortOnDevice(DATA *arr, size_t size);
__global__ void gpu_mergesort(DATA *A, DATA *B, size_t size, size_t width);
__device__ void gpu_bottomUpMerge(DATA *arr1, size_t size1, DATA *arr2,
                                  size_t size2, DATA *tmp);
__global__ void gpu_mergesort_tasksize(DATA *arr, DATA *tmp, size_t size,
                                      size_t tasksize);
__device__ int gpu_serial_merge_sort(DATA *arr, DATA *tmp, size_t n);

int main(int argc, char **argv) {
  DATA *arr;
  size_t size = SIZE;
  assert(GRIDSIZE * BLOCKSIZE == SIZE / TASKSIZE);
  assert(size == 0 || !(size & (size - 1)));

  arr = (DATA *)malloc(size * sizeof(DATA));
  if (arr == NULL) {
    fprintf(stderr, "Memory could not be allocated");
    exit(EXIT_FAILURE);
  }

  srand(0);
  for (size_t i = 0; i < size; i++) {
    arr[i] = rand();  // TODO: generate with sign and maybe in a range
  }

  MergeSortOnDevice(arr, size);
  assert(_is_sorted(arr, size) == 1);
}

int _is_sorted(DATA *arr, size_t size) {
  for (size_t i = 0; i < size - 1; i++)
    if (arr[i] > arr[i + 1]) return 0;
  return 1;
}

void MergeSortOnDevice(DATA *arr, size_t size) {
  if (size == 0) return;

  DATA *dArr, *tmp;

  size_t byteSize = size * sizeof(DATA);
  CUDA_CHECK(hipMalloc(&dArr, byteSize));
  CUDA_CHECK(hipMemcpy(dArr, arr, byteSize, hipMemcpyHostToDevice));
  CUDA_CHECK(hipMalloc(&tmp, byteSize));

  hipEvent_t start, stop;
  CUDA_CHECK(hipEventCreate(&start));
  CUDA_CHECK(hipEventCreate(&stop));
  CUDA_CHECK(hipEventRecord(start, 0));

  //
  // Slice up the list and give pieces of it to each thread, letting the pieces
  // grow bigger and bigger until the whole list is sorted
  //

  DATA *A = dArr, *B = tmp;

  int nBlocks = GRIDSIZE;
  int blockSize = BLOCKSIZE;
  gpu_mergesort_tasksize<<<nBlocks, blockSize>>>(A, B, size, TASKSIZE);

  for (size_t width = TASKSIZE; width <= size; width <<= 1) {
    // int slices = size / (nBlocks * blockSize * width);

    // Actually call the kernel
    gpu_mergesort<<<nBlocks, blockSize>>>(A, B, size, width);
    // gpu_mergesort<<<nBlocks, nThreads / blocksPerGrid>>>(
    //     A, B, size, width, slices, D_threads, D_blocks);

    // Switch the input / output arrays instead of copying them around
    A = A == dArr ? tmp : dArr;
    B = B == dArr ? tmp : dArr;

    if (blockSize > 1) {
      blockSize /= 2;
    } else {
      nBlocks /= 2;
    }
  }

  // MergeSortKernel<<<numBlocks, numThreads>>>(
  //     dArr, size, ceil(size / float(BLOCKSIZE * GRIDSIZE)));

  CUDA_CHECK(hipEventRecord(stop, 0));
  CUDA_CHECK(hipEventSynchronize(stop));
  float elapsed;
  CUDA_CHECK(hipEventElapsedTime(&elapsed, start, stop));
  elapsed = elapsed / 1000.f;  // convert to seconds
  CUDA_CHECK(hipEventDestroy(start));
  CUDA_CHECK(hipEventDestroy(stop));

  printf("Kernel elapsed time %fs \n", elapsed);

  CUDA_CHECK(hipMemcpy(arr, A, byteSize, hipMemcpyDeviceToHost));

  CUDA_CHECK(hipFree(dArr));
}

__global__ void gpu_mergesort_tasksize(DATA *arr, DATA *tmp, size_t size,
                                       size_t tasksize) {
  DATA *A = arr, *B = tmp;
  int n_swaps;
  size_t start = tasksize * (blockIdx.x * blockDim.x + threadIdx.x);
  if (start >= size) return;

  n_swaps = gpu_serial_merge_sort(arr + start, tmp + start, tasksize);
  if (n_swaps % 2 == 0) return;
  memcpy(arr + start, tmp + start, tasksize * sizeof(DATA));
}

__global__ void gpu_mergesort(DATA *A, DATA *B, size_t size, size_t width) {
  size_t start = width * (blockIdx.x * blockDim.x + threadIdx.x);

  if (start >= size) return;

  size_t halfSize = width / 2;

  gpu_bottomUpMerge(A + start, halfSize, A + start + halfSize, halfSize,
                    B + start);
}

__device__ int gpu_serial_merge_sort(DATA *arr, DATA *tmp, size_t n) {
  if (n == 0) return;
  // print_array(arr, n);
  // printf(
  //     "------------------------------------------------------------------\n");
  // print_array(tmp, n);
  // printf(
  //     "------------------------------------------------------------------\n");

  int n_swaps = 0;
  DATA *A = arr, *B = tmp;

  for (size_t curr_size = 1; curr_size <= n - 1; curr_size *= 2) {
    for (size_t left_start = 0; left_start <= n - curr_size - 1;
         left_start += 2 * curr_size) {
      // int left_size = MIN(curr_size, n - left_start);
      int right_size = MIN(curr_size, n - left_start - curr_size);

      // if (left_size < curr_size) break;
      gpu_bottomUpMerge(A + left_start, curr_size, A + left_start + curr_size,
                        // right_size, B);
                        curr_size, B + left_start);
    }
    A = A == arr ? tmp : arr;
    B = B == arr ? tmp : arr;
    n_swaps++;
    // print_array(A, n);
    // printf(
    //     "------------------------------------------------------------------\n");
  }
  return n_swaps;
}

__device__ void gpu_bottomUpMerge(DATA *arr1, size_t size1, DATA *arr2,
                                  size_t size2, DATA *tmp) {
  size_t i = 0, j = 0;

  while (i < size1 && j < size2) {
    if (arr1[i] < arr2[j]) {
      tmp[i + j] = arr1[i];
      i++;
    } else {
      tmp[i + j] = arr2[j];
      j++;
    }
  }
  while (i < size1) {
    tmp[i + j] = arr1[i];
    i++;
  }
  while (j < size2) {
    tmp[i + j] = arr2[j];
    j++;
  }
}